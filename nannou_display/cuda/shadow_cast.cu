#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include "shadow_cast.hpp"

#define PREPROCESS_BLOCK 4

float* point_angles = nullptr, *sorted_angles = nullptr;
bool* next_valid = nullptr;
int all_point_num = 0;              // set in memAllocator

__constant__ Vec2 all_points[4096];
__constant__ char next_ids[4096];

__host__ void deallocatePoints() {
    CUDA_CHECK_RETURN(hipFree(point_angles));
    CUDA_CHECK_RETURN(hipFree(sorted_angles));
    CUDA_CHECK_RETURN(hipFree(next_valid));
}

__host__ void updatePointInfo(const Vec2* const meshes, const char* const nexts, int point_num, bool initialized) {
    size_t mesh_point_cnt = 0;
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(all_points), meshes, sizeof(Vec2) * point_num, 0, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(next_ids), nexts, sizeof(char) * point_num, 0, hipMemcpyHostToDevice));
    if (initialized == true)
        deallocatePoints();
    CUDA_CHECK_RETURN(hipMalloc((void**) &point_angles, sizeof(float) * point_num));
    CUDA_CHECK_RETURN(hipMalloc((void**) &sorted_angles, sizeof(float) * point_num));
    CUDA_CHECK_RETURN(hipMalloc((void**) &next_valid, sizeof(bool) * point_num));
    all_point_num = point_num;
}


__host__ void shadowCasting(const Vec3& pose, Vec2* const host_output, int& point_num) {
    const int thread_per_block = static_cast<int>(std::ceil(static_cast<float>(all_point_num) / PREPROCESS_BLOCK));
    backCullPreprocessKernel<<< PREPROCESS_BLOCK, thread_per_block >>> (all_points, next_ids, pose, all_point_num, point_angles, next_valid);
    // sorting rays
    CUDA_CHECK_RETURN(hipMemcpy(sorted_angles, point_angles, all_point_num * sizeof(float), hipMemcpyDeviceToDevice));
    thrust::sort(thrust::device, sorted_angles, sorted_angles + all_point_num, thrust::less<float>());
    const int invalid_bound = thrust::lower_bound(thrust::device, sorted_angles, sorted_angles + all_point_num, 1e2, thrust::less<float>()) - sorted_angles;
    /// duplicate valid rays
    float* actual_rays = nullptr, *output_depth = nullptr;
    Vec2* out_pts = nullptr;
    const int actual_ray_num = invalid_bound << 1;              // double the number of valid rays (left & right extensions)
    const size_t ray_size = sizeof(float) * actual_ray_num;
    CUDA_CHECK_RETURN(hipMalloc((void **) &actual_rays, sizeof(float) * actual_ray_num));
    CUDA_CHECK_RETURN(hipMalloc((void **) &out_pts, sizeof(Vec2) * actual_ray_num));
    CUDA_CHECK_RETURN(hipMalloc((void **) &output_depth, ray_size << 2));
    simpleDuplicateKernel<<< 1, invalid_bound >>> (sorted_angles, actual_rays);
    /// get ray - mesh segment intersections. Notice that point_num (all_point_num) equals number of segment
    size_t ray_each_block = static_cast<size_t>(std::ceil(static_cast<float>(actual_ray_num) / 8));
    size_t seg_each_block = static_cast<size_t>(std::ceil(static_cast<float>(all_point_num) / 4));
    const size_t shared_memory_size = sizeof(int) * ray_each_block;
    hipStream_t streams[4];
    for (short i = 0; i < 4; i++)
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    for (int i = 0; i < 2; i++) {				// 面片
        for (int j = 0; j < 4; j++) {			// 光线
            // pose 在本处是const Vec3&, 在进入kernel时会发生复制，可以吗？
            pointIntersectKernel<<<dim3(2, 2), dim3(ray_each_block, seg_each_block), shared_memory_size, streams[j]>>>(
                all_points, next_ids, actual_rays, point_angles, next_valid, output_depth, pose, all_point_num, actual_ray_num, j, i
            );
        }
    }
    /// output, cleaning up
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    depth2PointKernel<<<1, invalid_bound>>>(output_depth, actual_rays, pose, out_pts);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipMemcpy(host_output, out_pts, sizeof(Vec2) * actual_ray_num, hipMemcpyDeviceToHost));
    for (int i = 0; i < 4; i++)
        hipStreamDestroy(streams[i]);
    CUDA_CHECK_RETURN(hipFree(out_pts));
    CUDA_CHECK_RETURN(hipFree(actual_rays));
    point_num = actual_ray_num;
}
